#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <chrono>
#include "kernel.h"

using namespace std;

#define KERNEL_SIZE 3

__global__ void convolutionKernel(int* f, const int* c, int* v, int width, int height)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < height && j < width)
    {
        int result = 0;

        for (int m = 0; m < KERNEL_SIZE; ++m)
        {
            for (int n = 0; n < KERNEL_SIZE; ++n)
            {
                int row = i + m - 1;
                int col = j + n - 1;

                // Check boundaries
                if (row >= 0 && row < height && col >= 0 && col < width)
                {
                    result += f[row * width + col] * c[m * KERNEL_SIZE + n];
                }
                else
                {
                    // Handle border conditions
                    if (row == -1)
                        row = 0;
                    else if (row == height)
                        row = height - 1;

                    if (col == -1)
                        col = 0;
                    else if (col == width)
                        col = width - 1;

                    result += f[row * width + col] * c[m * KERNEL_SIZE + n];
                }
            }
        }
        printf("i:%d j:%d result:%d\n", i, j, result);
        v[i * width + j] = result;
    }
}

void generateRandomMatrixToFile(int n, int m, int minValue, int maxValue, const string& fileName) {

    srand(time(0));

    try {
        ofstream file(fileName);

        if (file.is_open()) {

            file << n << " " << m << "\n";


            for (int i = 0; i < n; i++) {
                for (int j = 0; j < m; j++) {
                    int randomValue = rand() % (maxValue - minValue + 1) + minValue;
                    file << randomValue << " ";
                }
                file << "\n";
            }

            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 3; j++) {
                    file << rand() % 2 << " ";
                }
                file << "\n";
            }

            file.close();
        }
        else {
            cerr << "Unable to open the file: " << fileName << endl;
        }
    }
    catch (const exception& e) {
        cerr << "Error: " << e.what() << endl;
    }
}

void readMatrixFromFile(const char* filename, int*& matrix, int*& result,int* kernel, int& width, int& height)
{
    FILE* file = fopen(filename, "r");
    if (!file)
    {
        printf("Error opening file.\n");
        exit(EXIT_FAILURE);
    }

    fscanf(file, "%d %d", &width, &height);

    matrix = new int[width * height];
    result = new int[width * height];

    for (int i = 0; i < height; ++i)
    {
        for (int j = 0; j < width; ++j)
        {
            fscanf(file, "%d", &matrix[i * width + j]);
        }
    }

    for (int i = 0; i < KERNEL_SIZE; ++i)
    {
        for (int j = 0; j < KERNEL_SIZE; ++j)
        {
            fscanf(file, "%d", &kernel[i * KERNEL_SIZE + j]);
        }
    }

    fclose(file);
}

void writeResultToFile(int height, int width, int* v)
{

    std::ofstream outFile("output.txt");
    if (!outFile.is_open())
    {
        std::cerr << "Error opening file for writing." << std::endl;
        return;
    }

    for (int i = 0; i < height; ++i)
    {
        for (int j = 0; j < width; ++j)
        {
            outFile << v[i * width + j] << " ";
        }
        outFile << std::endl;
    }

    outFile.close();

}

int main()
{   
    int width, height;
    int* f;
    // Result matrix
    int* v;
    int c[KERNEL_SIZE * KERNEL_SIZE];

    //generateRandomMatrixToFile(10, 10, 0, 10, "data1.txt");

    // Read the matrix from the file
    readMatrixFromFile("input_1.txt", f, v, c, width, height);

    int* dev_f, * dev_c, * dev_v;

    // Allocate device memory
    hipMalloc((void**)&dev_f, width * height * sizeof(int));
    hipMalloc((void**)&dev_c, KERNEL_SIZE * KERNEL_SIZE * sizeof(int));
    hipMalloc((void**)&dev_v, width * height * sizeof(int));

    // Copy data from host to device
    hipMemcpy(dev_f, f, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, KERNEL_SIZE * KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 blockSize(16, 16); //2x2 threads per block
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    auto time1 = chrono::steady_clock::now();

    // Launch the CUDA kernel
    convolutionKernel <<<gridSize, blockSize>>> (dev_f, dev_c, dev_v, width, height);

    // Copy the result back to the host
    hipMemcpy(v, dev_v, width * height * sizeof(int), hipMemcpyDeviceToHost);

    auto time2 = chrono::steady_clock::now();
    auto diff = time2 - time1;
    cout << chrono::duration <double, milli>(diff).count();

    // Clean up
    hipFree(dev_f);
    hipFree(dev_c);
    hipFree(dev_v);

    writeResultToFile(height, width, v);

    delete[] f;
    delete[] v;

    

    return 0;
}


